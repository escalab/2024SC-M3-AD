#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <cublas.h>
#include <hipblas.h>

#define BLOCK_DIM 16



/**
 * For each reference point (i.e. each column) finds the k-th smallest distances
 * of the distance matrix and their respective indexes and gathers them at the top
 * of the 2 arrays.
 *
 * Since we only need to locate the k smallest distances, sorting the entire array
 * would not be very efficient if k is relatively small. Instead, we perform a
 * simple insertion sort by eventually inserting a given distance in the first
 * k values.
 *
 * @param dist         distance matrix
 * @param dist_pitch   pitch of the distance matrix given in number of columns
 * @param index        index matrix
 * @param index_pitch  pitch of the index matrix given in number of columns
 * @param width        width of the distance matrix and of the index matrix
 * @param height       height of the distance matrix
 * @param k            number of values to find
 */
__global__ void modified_insertion_sort(float * dist,
                                        int     dist_pitch,
                                        int *   index,
                                        int     index_pitch,
                                        int     width,
                                        int     height,
                                        int     k){

    // Column position
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Do nothing if we are out of bounds
    if (xIndex < width) {

        // Pointer shift
        float * p_dist  = dist  + xIndex;
        int *   p_index = index + xIndex;

        // Initialise the first index
        p_index[0] = 0;

        // Go through all points
        for (int i=1; i<height; ++i) {

            // Store current distance and associated index
            float curr_dist = p_dist[i*dist_pitch];
            int   curr_index  = i;

            // Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
            if (i >= k && curr_dist >= p_dist[(k-1)*dist_pitch]) {
                continue;
            }

            // Shift values (and indexes) higher that the current distance to the right
            int j = min(i, k-1);
            while (j > 0 && p_dist[(j-1)*dist_pitch] > curr_dist) {
                p_dist[j*dist_pitch]   = p_dist[(j-1)*dist_pitch];
                p_index[j*index_pitch] = p_index[(j-1)*index_pitch];
                --j;
            }

            // Write the current distance and index at their position
            p_dist[j*dist_pitch]   = curr_dist;
            p_index[j*index_pitch] = curr_index; 
        }
    }
}



/**
 * Computes the squared norm of each column of the input array.
 *
 * @param array   input array
 * @param width   number of columns of `array` = number of points
 * @param pitch   pitch of `array` in number of columns
 * @param height  number of rows of `array` = dimension of the points
 * @param norm    output array containing the squared norm values
 */
__global__ void compute_squared_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (xIndex<width){
        float sum = 0.f;
        for (int i=0; i<height; i++){
            float val = array[i*pitch+xIndex];
            sum += val*val;
        }
        norm[xIndex] = sum;
    }
}


/**
 * Add the reference points norm (column vector) to each colum of the input array.
 *
 * @param array   input array
 * @param width   number of columns of `array` = number of points
 * @param pitch   pitch of `array` in number of columns
 * @param height  number of rows of `array` = dimension of the points
 * @param norm    reference points norm stored as a column vector
 */
__global__ void add_reference_points_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int xIndex = blockIdx.x * blockDim.x + tx;
    unsigned int yIndex = blockIdx.y * blockDim.y + ty;
    __shared__ float shared_vec[16];
    if (tx==0 && yIndex<height)
        shared_vec[ty] = norm[yIndex];
    __syncthreads();
    if (xIndex<width && yIndex<height)
        array[yIndex*pitch+xIndex] += shared_vec[ty];
}


/**
 * Adds the query points norm (row vector) to the k first lines of the input
 * array and computes the square root of the resulting values.
 *
 * @param array   input array
 * @param width   number of columns of `array` = number of points
 * @param pitch   pitch of `array` in number of columns
 * @param k       number of neighbors to consider
 * @param norm     query points norm stored as a row vector
 */
__global__ void add_query_points_norm_and_sqrt(float * array, int width, int pitch, int k, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        array[yIndex*pitch + xIndex] = sqrt(array[yIndex*pitch + xIndex] + norm[xIndex]);
}


bool knn_cublas(const float * ref,
                int           ref_nb,
                const float * query,
                int           query_nb,
                int           dim, 
                int           k, 
                float *       knn_dist,
                int *         knn_index) {

    // Constants
    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);

    // Return variables
    hipError_t  err0, err1, err2, err3, err4, err5;

    // Check that we have at least one CUDA device 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    // Initialize CUBLAS
    // cublasInit();
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH);
    // cublasSetMathMode(cublasHandle, CUBLAS_TENSOR_OP_MATH));

    // Allocate global memory
    float * ref_dev        = NULL;
    float * query_dev      = NULL;
    float * dist_dev       = NULL;
    int   * index_dev      = NULL;
    float * ref_norm_dev   = NULL;
    float * query_norm_dev = NULL;
    size_t  ref_pitch_in_bytes;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&ref_dev,   &ref_pitch_in_bytes,   ref_nb   * size_of_float, dim);
    err1 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err2 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err3 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    err4 = hipMalloc((void**)&ref_norm_dev,   ref_nb   * size_of_float);
    err5 = hipMalloc((void**)&query_norm_dev, query_nb * size_of_float);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess) {
        printf("ERROR: Memory allocation error\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        // cublasShutdown();
        return false;
    }

    // Deduce pitch values
    size_t ref_pitch   = ref_pitch_in_bytes   / size_of_float;
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;

    // Check pitch values
    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        // cublasShutdown();
        return false; 
    }

    // Copy reference and query data from the host to the device
    err0 = hipMemcpy2D(ref_dev,   ref_pitch_in_bytes,   ref,   ref_nb * size_of_float,   ref_nb * size_of_float,   dim, hipMemcpyHostToDevice);
    err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        printf("%s\n", hipGetErrorString(err0));
        printf("%s\n", hipGetErrorString(err1));
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        // cublasShutdown();
        return false; 
    }

    // Compute the squared norm of the reference points
    dim3 block0(256, 1, 1);
    dim3 grid0(ref_nb / 256, 1, 1);
    if (ref_nb % 256 != 0) grid0.x += 1;
    compute_squared_norm<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, dim, ref_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        // cublasShutdown();
        return false;
    }

    // Compute the squared norm of the query points
    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    compute_squared_norm<<<grid1, block1>>>(query_dev, query_nb, query_pitch, dim, query_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        // cublasShutdown();
        return false;
    }

    // Computation of query*transpose(reference)
    // cublasSgemm('n', 't', (int)query_pitch, (int)ref_pitch, dim, (float)-2.0, query_dev, query_pitch, ref_dev, ref_pitch, (float)0.0, dist_dev, query_pitch);
    float alpha = -2.0f;
    float beta = 0.0f;
    // cublasSgemm(cublasHandle, CUBLAS_OP_N, CUBLAS_OP_T, 
    //     (int)query_pitch, (int)ref_pitch, dim, 
    //     &alpha,
    //     query_dev, query_pitch,
    //     ref_dev, ref_pitch,
    //     &beta, 
    //     dist_dev,query_pitch);

    hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
        (int)query_pitch, (int)ref_pitch, (int)dim, 
        &alpha,
        query_dev, HIP_R_32F,query_pitch,
        ref_dev, HIP_R_32F,ref_pitch,
        &beta,
        dist_dev, HIP_R_32F, query_pitch,
        HIPBLAS_COMPUTE_32F_FAST_TF32,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP 
    );
    hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
        (int)query_pitch, (int)ref_pitch, (int)dim, 
        &alpha,
        query_dev, HIP_R_32F,query_pitch,
        ref_dev, HIP_R_32F,ref_pitch,
        &beta,
        dist_dev, HIP_R_32F, query_pitch,
        HIPBLAS_COMPUTE_32F_FAST_TF32,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP 
    );
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        hipblasDestroy(cublasHandle);
        // cublasShutdown();
        return false;
    }

    // Add reference points norm
    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, ref_nb / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (ref_nb   % 16 != 0) grid2.y += 1;
    add_reference_points_norm<<<grid2, block2>>>(dist_dev, query_nb, dist_pitch, ref_nb, ref_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        hipblasDestroy(cublasHandle);
        // cublasShutdown();
        return false;
    }

    // Sort each column
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        hipblasDestroy(cublasHandle);
        // cublasShutdown();
        return false;
    }

    // Add query norm and compute the square root of the of the k first elements
    dim3 block3(16, 16, 1);
    dim3 grid3(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid3.x += 1;
    if (k        % 16 != 0) grid3.y += 1;
    add_query_points_norm_and_sqrt<<<grid3, block3>>>(dist_dev, query_nb, dist_pitch, k, query_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        hipblasDestroy(cublasHandle);
        // cublasShutdown();
        return false;
    }

    // Copy k smallest distances / indexes from the device to the host
    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        hipblasDestroy(cublasHandle);
        // cublasShutdown();
        return false; 
    }

    // Memory clean-up and CUBLAS shutdown
    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev);
    hipFree(ref_norm_dev);
    hipFree(query_norm_dev);
    hipblasDestroy(cublasHandle);
    // cublasShutdown();

    return true;
}
