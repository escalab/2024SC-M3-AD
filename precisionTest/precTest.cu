#include "hip/hip_runtime.h"
#include "precTest.hpp"

// random float number between -1 and 1
#define randf() 2*( (double)rand() / (double)RAND_MAX )-1

// Detect errors
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess)
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
}
#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS)
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
}

int main(int argc, char** argv )
{
    if ( argc != 2 ) {
        printf("usage: ./precTest [size]\n");
        return -1;
    }

    printf("SIZE: %s\n", argv[1]);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // set sizes
    int M = atoi(argv[1]);
    int N = atoi(argv[1]);
    int K = atoi(argv[1]);

    double *h_A, *h_B;
    double *h_C, *h_D;
    hipHostMalloc(&h_A, M * K * sizeof(double));
    hipHostMalloc(&h_B, K * N * sizeof(double));
    hipHostMalloc(&h_C, M * N * sizeof(double));
    hipHostMalloc(&h_D, M * N * sizeof(double));

    // initialize h_A and h_B
    srand((unsigned)time(NULL));
    for (int i = 0; i < M * K; i++)
        h_A[i] = randf();
    for (int i = 0; i < K * N; i++)
        h_B[i] = randf();
    for (int i = 0; i < M * N; i++) {
        h_C[i] = randf();
        h_D[i] = 0;
    }

    // print h_A, h_B, and h_C
    // std::cout << std::endl;
    // for(int i=0; i<M; i++) {
    //     for(int j=0; j<K; j++) {
    //         std::cout << (double)h_A[i*K+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;
    // for(int i=0; i<K; i++) {
    //     for(int j=0; j<N; j++) {
    //         std::cout << (double)h_B[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;
    // for(int i=0; i<M; i++) {
    //     for(int j=0; j<N; j++) {
    //         std::cout << (double)h_C[j*M+i] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;

    // copy h_A and h_B to device
    double *d_A;
    hipMalloc(&d_A, M * K * sizeof(double));
    hipMemcpy(d_A, h_A, M * K * sizeof(double), hipMemcpyHostToDevice);
    
    double *d_B;
    hipMalloc(&d_B, K * N * sizeof(double));
    hipMemcpy(d_B, h_B, K * N * sizeof(double), hipMemcpyHostToDevice);
    
    double *d_C;
    hipMalloc(&d_C, M * N * sizeof(double));
    hipMemcpy(d_C, h_C, M * N * sizeof(double), hipMemcpyHostToDevice);

    // launch kernel
    const double alphaD = 1;
    const double betaD = 1;

/// FP64 //////////////////////////////////////////////////////////////////////
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                                &alphaD, 
                                d_A, HIP_R_64F, K, 
                                d_B, HIP_R_64F, N, 
                                &betaD, 
                                d_C, HIP_R_64F, M, 
                                HIPBLAS_COMPUTE_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );

    hipMemcpy(h_D, d_C, M * N * sizeof(double), hipMemcpyDeviceToHost);

    // C is transposed
    // for(int i=0; i<M; i++) {
    //     for(int j=0; j<N; j++) {
    //         std::cout << (double)h_C[j*M+i] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;

/// for different data types //////////////////////////////////////////////////

    const float alphaF = 1;
    const float betaF = 1;

    float *h_A1, *h_B1;
    float *h_C1, *h_D1;
    hipHostMalloc(&h_A1, M * K * sizeof(float));
    hipHostMalloc(&h_B1, K * N * sizeof(float));
    hipHostMalloc(&h_C1, M * N * sizeof(float));
    hipHostMalloc(&h_D1, M * N * sizeof(float));

    for (int i = 0; i < M * K; i++)
        h_A1[i] = (float)h_A[i];
    for (int i = 0; i < K * N; i++)
        h_B1[i] = (float)h_B[i];
    for (int i = 0; i < M * N; i++) {
        h_C1[i] = (float)h_C[i];
        h_D1[i] = 0;
    }

    // copy h_A and h_B to device
    float *d_A1;
    hipMalloc(&d_A1, M * K * sizeof(float));
    hipMemcpy(d_A1, h_A1, M * K * sizeof(float), hipMemcpyHostToDevice);
    
    float *d_B1;
    hipMalloc(&d_B1, K * N * sizeof(float));
    hipMemcpy(d_B1, h_B1, K * N * sizeof(float), hipMemcpyHostToDevice);
    
    float *d_C1;
    hipMalloc(&d_C1, M * N * sizeof(float));

    std::cout<<std::fixed;
    std::cout.precision(15);

/// FP32 = 3BP16 (Ma) = 3TF32 (cuTlass) ///////////////////////////////////////
    hipMemcpy(d_C1, h_C1, M * N * sizeof(float), hipMemcpyHostToDevice);
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_A1, HIP_R_32F, K, 
                            d_B1, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );

    hipMemcpy(h_D1, d_C1, M * N * sizeof(float), hipMemcpyDeviceToHost);

    double err = 0;
    for (int i=0; i<M*N; i++) {
        if(err < fabs((double)h_D1[i] - h_D[i]))
            err = fabs((double)h_D1[i] - h_D[i]);
    }
    std::cout << "FP32 error: \t" << err << std::endl;

    hipMemcpy(d_C1, h_C1, M * N * sizeof(float), hipMemcpyHostToDevice);

/// FP16 //////////////////////////////////////////////////////////////////////
    hipMemcpy(d_C1, h_C1, M * N * sizeof(float), hipMemcpyHostToDevice);
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_A1, HIP_R_32F, K, 
                            d_B1, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );

    hipMemcpy(h_D1, d_C1, M * N * sizeof(float), hipMemcpyDeviceToHost);

    err = 0;
    for (int i=0; i<M*N; i++) {
        if(err < fabs((double)h_D1[i] - h_D[i]))
            err = fabs((double)h_D1[i] - h_D[i]);
    }
    std::cout << "FP16 error: \t" << err << std::endl;

/// BP16 //////////////////////////////////////////////////////////////////////
    hipMemcpy(d_C1, h_C1, M * N * sizeof(float), hipMemcpyHostToDevice);
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_A1, HIP_R_32F, K, 
                            d_B1, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_16BF, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );

    hipMemcpy(h_D1, d_C1, M * N * sizeof(float), hipMemcpyDeviceToHost);

    err = 0;
    for (int i=0; i<M*N; i++) {
        if(err < fabs((double)h_D1[i] - h_D[i]))
            err = fabs((double)h_D1[i] - h_D[i]);
    }
    std::cout << "BFP16 error: \t" << err << std::endl;

/// TF32 //////////////////////////////////////////////////////////////////////
    hipMemcpy(d_C1, h_C1, M * N * sizeof(float), hipMemcpyHostToDevice);
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_A1, HIP_R_32F, K, 
                            d_B1, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_TF32, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );

    hipMemcpy(h_D1, d_C1, M * N * sizeof(float), hipMemcpyDeviceToHost);

    err = 0;
    for (int i=0; i<M*N; i++) {
        if(err < fabs((double)h_D1[i] - h_D[i]))
            err = fabs((double)h_D1[i] - h_D[i]);
    }
    std::cout << "TF32 error: \t" << err << std::endl;

    hipFree(d_A1);
    hipFree(d_B1);
    hipFree(d_C1);
    hipHostFree(h_A1);
    hipHostFree(h_B1);
    hipHostFree(h_C1);
    hipHostFree(h_D1);

/// 2FP16 (Markidis) //////////////////////////////////////////////////////////

    // half *h_AH1, *h_AL1, *h_BH1, *h_BL1;
    // hipHostMalloc(&h_AH1, M * K * sizeof(half));
    // hipHostMalloc(&h_AL1, M * K * sizeof(half));
    // hipHostMalloc(&h_BH1, K * N * sizeof(half));
    // hipHostMalloc(&h_BL1, K * N * sizeof(half));
    // hipHostMalloc(&h_C1, M * N * sizeof(float));
    // hipHostMalloc(&h_D1, M * N * sizeof(float));

    // for (int i = 0; i < M * K; i++) {
    //     h_AH1[i] = (half)h_A[i];
    //     h_AL1[i] = (half)(h_A[i] - (float)(h_AH1[i]));
    // }
    // for (int i = 0; i < K * N; i++) {
    //     h_BH1[i] = (half)h_B[i];
    //     h_BL1[i] = (half)(h_B[i] - (float)(h_BH1[i]));
    // }
    // for (int i = 0; i < M * N; i++) {
    //     h_C1[i] = (float)h_C[i];
    //     h_D1[i] = 0;
    // }

    // // copy h_A and h_B to device
    // half *d_AH1;
    // hipMalloc(&d_AH1, M * K * sizeof(half));
    // hipMemcpy(d_AH1, h_AH1, M * K * sizeof(half), hipMemcpyHostToDevice);

    // half *d_AL1;
    // hipMalloc(&d_AL1, M * K * sizeof(half));
    // hipMemcpy(d_AL1, h_AL1, M * K * sizeof(half), hipMemcpyHostToDevice);
    
    // half *d_BH1;
    // hipMalloc(&d_BH1, K * N * sizeof(half));
    // hipMemcpy(d_BH1, h_BH1, K * N * sizeof(half), hipMemcpyHostToDevice);

    // half *d_BL1;
    // hipMalloc(&d_BL1, K * N * sizeof(half));
    // hipMemcpy(d_BL1, h_BL1, K * N * sizeof(half), hipMemcpyHostToDevice);
    
    // hipMalloc(&d_C1, M * N * sizeof(float));
    // hipMemcpy(d_C1, h_C1, M * N * sizeof(float), hipMemcpyHostToDevice);

    // cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
    //                         &alphaF, 
    //                         d_AH1, HIP_R_16F, K, 
    //                         d_BH1, HIP_R_16F, N, 
    //                         &betaF, 
    //                         d_C1, HIP_R_32F, M, 
    //                         HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );
    // cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
    //                         &alphaF, 
    //                         d_AH1, HIP_R_16F, K, 
    //                         d_BL1, HIP_R_16F, N, 
    //                         &betaF, 
    //                         d_C1, HIP_R_32F, M, 
    //                         HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );
    // cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
    //                         &alphaF, 
    //                         d_AL1, HIP_R_16F, K, 
    //                         d_BH1, HIP_R_16F, N, 
    //                         &betaF, 
    //                         d_C1, HIP_R_32F, M, 
    //                         HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );
    // cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
    //                         &alphaF, 
    //                         d_AL1, HIP_R_16F, K, 
    //                         d_BL1, HIP_R_16F, N, 
    //                         &betaF, 
    //                         d_C1, HIP_R_32F, M, 
    //                         HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );

    // hipMemcpy(h_D1, d_C1, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // err = 0;
    // for (int i=0; i<M*N; i++) {
    //     if(err < fabs((double)h_D1[i] - h_D[i]))
    //         err = fabs((double)h_D1[i] - h_D[i]);
    // }
    // std::cout << "2FP16 error: \t" << err << std::endl;

    // hipFree(d_AH1);
    // hipFree(d_AL1);
    // hipFree(d_BH1);
    // hipFree(d_BL1);
    // hipFree(d_C1);
    // hipHostFree(h_AH1);
    // hipHostFree(h_AL1);
    // hipHostFree(h_BH1);
    // hipHostFree(h_BL1);
    // hipHostFree(h_C1);
    // hipHostFree(h_D1);

/// 2TF32 (Ma) ////////////////////////////////////////////////////////////////

    float *h_AH2, *h_AL2, *h_BH2, *h_BL2;
    hipHostMalloc(&h_AH2, M * K * sizeof(float));
    hipHostMalloc(&h_AL2, M * K * sizeof(float));
    hipHostMalloc(&h_BH2, K * N * sizeof(float));
    hipHostMalloc(&h_BL2, K * N * sizeof(float));
    hipHostMalloc(&h_C1, M * N * sizeof(float));
    hipHostMalloc(&h_D1, M * N * sizeof(float));

    for (int i = 0; i < M * K; i++) {
        h_AH2[i] = (half)h_A[i];
        h_AL2[i] = (half)(h_A[i] - h_AH2[i]);
    }
    for (int i = 0; i < K * N; i++) {
        h_BH2[i] = (half)h_B[i];
        h_BL2[i] = (half)(h_B[i] - h_BH2[i]);
    }
    for (int i = 0; i < M * N; i++) {
        h_C1[i] = (float)h_C[i];
        h_D1[i] = 0;
    }

    // copy h_A and h_B to device
    float *d_AH2;
    hipMalloc(&d_AH2, M * K * sizeof(float));
    hipMemcpy(d_AH2, h_AH2, M * K * sizeof(float), hipMemcpyHostToDevice);

    float *d_AL2;
    hipMalloc(&d_AL2, M * K * sizeof(float));
    hipMemcpy(d_AL2, h_AL2, M * K * sizeof(float), hipMemcpyHostToDevice);
    
    float *d_BH2;
    hipMalloc(&d_BH2, K * N * sizeof(float));
    hipMemcpy(d_BH2, h_BH2, K * N * sizeof(float), hipMemcpyHostToDevice);

    float *d_BL2;
    hipMalloc(&d_BL2, K * N * sizeof(float));
    hipMemcpy(d_BL2, h_BL2, K * N * sizeof(float), hipMemcpyHostToDevice);
    
    hipMalloc(&d_C1, M * N * sizeof(float));
    hipMemcpy(d_C1, h_C1, M * N * sizeof(float), hipMemcpyHostToDevice);

    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_AH2, HIP_R_32F, K, 
                            d_BH2, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_AH2, HIP_R_32F, K, 
                            d_BL2, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_AL2, HIP_R_32F, K, 
                            d_BH2, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_AL2, HIP_R_32F, K, 
                            d_BL2, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );

    hipMemcpy(h_D1, d_C1, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // err = 0;
    // for (int i=0; i<M*N; i++) {
    //     if(err < fabs((double)h_D1[i] - h_D[i]))
    //         err = fabs((double)h_D1[i] - h_D[i]);
    // }
    // std::cout << "2FP16 error: \t" << err << std::endl;

    hipMemcpy(d_C1, h_C1, M * N * sizeof(float), hipMemcpyHostToDevice);

    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_AH2, HIP_R_32F, K, 
                            d_BH2, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_TF32, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_AH2, HIP_R_32F, K, 
                            d_BL2, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_TF32, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_AL2, HIP_R_32F, K, 
                            d_BH2, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_TF32, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );
    cublasErrCheck( hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, 
                            &alphaF, 
                            d_AL2, HIP_R_32F, K, 
                            d_BL2, HIP_R_32F, N, 
                            &betaF, 
                            d_C1, HIP_R_32F, M, 
                            HIPBLAS_COMPUTE_32F_FAST_TF32, CUBLAS_GEMM_DEFAULT_TENSOR_OP) );

    hipMemcpy(h_D1, d_C1, M * N * sizeof(float), hipMemcpyDeviceToHost);

    err = 0;
    for (int i=0; i<M*N; i++) {
        if(err < fabs((double)h_D1[i] - h_D[i]))
            err = fabs((double)h_D1[i] - h_D[i]);
    }
    std::cout << "2TF32 error: \t" << err << std::endl;

    hipFree(d_AH2);
    hipFree(d_AL2);
    hipFree(d_BH2);
    hipFree(d_BL2);
    hipFree(d_C1);
    hipHostFree(h_AH2);
    hipHostFree(h_AL2);
    hipHostFree(h_BH2);
    hipHostFree(h_BL2);
    hipHostFree(h_C1);
    hipHostFree(h_D1);

    // destroy handle
    hipblasDestroy(handle);

    // Free the memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}